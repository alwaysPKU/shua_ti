#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <memory.h>
#include <fstream>
#include <sstream>

#include <vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <thrust/extrema.h>
#include <algorithm>

#include "hipblas.h"
#include "hip/hip_runtime.h"



#define Threshold 0.5

#define HASFN 1
#define HASLABEL 0

// #define NEEDNORMALIZATION 1
// #define MAXDIM 256
#define MAXPERBATCH 1000000
#define MAXSIZE 2000000
#define alloc_size 20000
#define IDX2C(i, j, ld) ((j)*(ld) +(i))
// #define MAXOUTPUTNUM 100000000
// #define unsigned long long LL
#define CHECK_CUBLAS_ERROR(state) if(HIPBLAS_STATUS_SUCCESS != state) \
printf("ERROR state %d in file %s at line %d.\n", state, __FILE__, __LINE__);


typedef  struct featsMat
{
     float* mat  = 0;
     int length = 0;
     int  dim = 0;
    
}featM;

struct saxpy_functor
{
    const int a;

    saxpy_functor(int _a) : a(_a) {}

    __host__ __device__
        thrust::pair    <float, int  > operator()(const float& x, const int& y) const { 
            return ( thrust::pair    <float,int> (x,y) );
        }
};

struct is_true
{
    const float threshold;

    is_true(float _threshold) : threshold(_threshold) {}

  __host__ __device__
  bool operator()( const thrust::pair    <float,int> &x )
  {
    // return (x.first>=Threshold)&&(x.first <0.9999);
    return x.first>= threshold;
  }
};


void normalizegpu( float* A, int l1,int dim) {
//not parallel
    float *dev_mat1 = 0, *norm = 0;
    CHECK_CUBLAS_ERROR(hipMalloc((void**)&dev_mat1, dim*sizeof(float)));
    norm = (float*)malloc(dim*sizeof(float));

    hipblasHandle_t matAddHandle;
    hipblasCreate(&matAddHandle);
    for ( int i=0;i<l1;i++){
            CHECK_CUBLAS_ERROR(hipblasSetVector(dim, sizeof(float), A+i*dim, 1, dev_mat1, 1));
            CHECK_CUBLAS_ERROR(hipblasSnrm2(matAddHandle,dim, dev_mat1, 1, norm)); 
            *norm=1/(*norm);
            CHECK_CUBLAS_ERROR(hipblasSscal(matAddHandle,dim, norm,dev_mat1,1)); 
            CHECK_CUBLAS_ERROR(hipblasGetVector( dim, sizeof(float),dev_mat1 , 1 , A+i*dim, 1));
    }

    CHECK_CUBLAS_ERROR(hipblasDestroy(matAddHandle));
    CHECK_CUBLAS_ERROR(hipFree(dev_mat1));
}



extern "C" void calcthreshold( float* A, float* B,int* paras,float threshold, int* index1,int* index2,float* scores) {
    //paras[int l1,int l2,int dim,int lvalue,int cnt,int gpuno  
//    hipSetDevice(paras[5]);
    clock_t starttime,endtime;
    starttime=clock();
    int l1=paras[0],l2=paras[1],dim=paras[2],lvalue=paras[3],cnt=paras[4],cntlimit=paras[5],cardno=paras[6];
    hipSetDevice(cardno);
    endtime=clock();
//    std::cout<<"set GPU"<<(double)(endtime-starttime)/CLOCKS_PER_SEC<<std::endl;


    float alpha = 1.0f;
    float beta = 0.0f;
    //FILE *stream; 
    //stream = fopen(fn, "w+");
    float *dev_mat1 = 0, *dev_mat2 = 0, *dev_y = 0;//, *seq = 0;//, *ttest = 0 ;//, *A_tmp, *B_tmp;
    thrust::pair    <float,int> *d_vector= 0, *true_vect = 0;
    int offset = 0,sum=0;
    int ceili=ceil((float)l2/alloc_size);
    int ceilj=ceil((float)l1/alloc_size);
    clock_t t1 = clock();
    int alloc_sizeA=min(l1,alloc_size);
    int alloc_sizeB=min(l2,alloc_size);
//    hipDeviceSetLimit(hipLimitMallocHeapSize, ((uint64_t)4*alloc_size*alloc_size)); 
    CHECK_CUBLAS_ERROR(hipMalloc((void**)&dev_mat1, (alloc_sizeA*dim)*sizeof(float)));
    CHECK_CUBLAS_ERROR(hipMalloc((void**)&dev_mat2, (alloc_sizeB*dim)*sizeof(float)));
    CHECK_CUBLAS_ERROR(hipMalloc((void**)&dev_y, (alloc_sizeA*alloc_sizeB)*sizeof(float)));
    CHECK_CUBLAS_ERROR(hipMalloc((void**)&d_vector, (alloc_sizeA*alloc_sizeB)*sizeof(thrust::pair    <float,int>)));
    CHECK_CUBLAS_ERROR(hipMalloc((void**)&true_vect, (alloc_sizeA*alloc_sizeB)*sizeof(thrust::pair    <float,int>)));
    

    clock_t t2 = clock();
  //  printf("malloc : %lf\n",(double)(t2-t1)/CLOCKS_PER_SEC);

    hipblasHandle_t matAddHandle;
    hipblasCreate(&matAddHandle);
    thrust::host_vector< thrust::pair    <float,int> > h_vector(MAXPERBATCH);
//    printf("l1=%d, l2=%d\n",l1,l2);   


    for ( int i=0;i<ceili;i++){//ceili
        int lb=min(l2-alloc_size*i,alloc_size);
        CHECK_CUBLAS_ERROR(hipblasSetMatrix(lb, dim, sizeof(float), B+(uint64_t)alloc_size*i*dim, lb, dev_mat2, lb));
        for ( int j=0;j<ceilj;j++){//ceilj
             clock_t t22 = clock();
            int la=min(l1-alloc_size*j,alloc_size);
           // printf("i=%d , j= %d , la=%d, lb=%d v=%f\n",i,j,la,lb,*A);
            CHECK_CUBLAS_ERROR(hipblasSetMatrix(la, dim, sizeof(float), A+(uint64_t)alloc_size*j*dim, la, dev_mat1, la));
           
            clock_t t3 = clock();
//             printf("set value: %lf\n",(double)(t3-t22)/CLOCKS_PER_SEC);
          
             clock_t t4 = clock();
            CHECK_CUBLAS_ERROR(hipblasSgemm(matAddHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, lb, la,dim,
                &alpha, dev_mat2, dim, dev_mat1, dim , &beta, dev_y,lb));
            
             clock_t t5 = clock();
//             printf("calculation : %lf  %d %d \n",(double)(t5-t3)/CLOCKS_PER_SEC, la, lb);

            thrust::device_ptr<float> iter( dev_y ); 
            thrust::device_ptr<thrust::pair <float,int> > dv( d_vector ); 
            thrust::device_ptr<thrust::pair <float,int> > tv( true_vect ); 

//            printf("now transform %d %d \n",i ,j );
            clock_t t6 = clock();
            thrust::transform(iter, iter+la*lb,  thrust::make_counting_iterator(0), dv, saxpy_functor(1));// i*ceilj +j may be dealt with later
//            printf("12321321313\n");
            offset = thrust::copy_if(dv, dv+la*lb, tv, is_true(threshold)) - tv ;
            sum+=offset;
//            printf("offset = %d\n",offset);
            int batches=(offset-1)/MAXPERBATCH+1;
            int pleft=0,pright=min(offset,MAXPERBATCH);
            while(batches--)
            {//   printf("%d %d %d\n",pleft,pright,batches);
                thrust::copy(tv+pleft,tv+pright,h_vector.begin());  
                for(int it=0;it<pright-pleft;it++)
                {
                    index1[paras[4]]=lvalue+j*alloc_size+h_vector[it].second/lb;
                    index2[paras[4]]=i*alloc_size+h_vector[it].second%lb;
                    scores[paras[4]]=h_vector[it].first;
                    paras[4]+=1;
                    if(paras[4]>=cntlimit)
                        return;
                    //fprintf(stream, "%d %d %f\n",  lvalue+j*alloc_size+h_vector[it].second/lb, i*alloc_size+h_vector[it].second%lb, h_vector[it].first);
                    
                }
                pleft=pright;
                pright+=min(MAXPERBATCH,offset-pleft);
                
            }
//            hipDeviceSynchronize();
            
        }
    }
    clock_t t3=clock();
//    printf("malloc : %lf\n",(double)(t3-t2)/CLOCKS_PER_SEC);
    //printf("l1=%d, l2=%d, dim=%d\n",l1,l2,dim);
    //printf("cardno:%d sum:%d\n",cardno,sum);
    //printf("%d %d %d %f\n",cardno,ceili,ceilj,*A);


    CHECK_CUBLAS_ERROR(hipblasDestroy(matAddHandle));
    CHECK_CUBLAS_ERROR(hipFree(dev_y));
    CHECK_CUBLAS_ERROR(hipFree(dev_mat1));
    CHECK_CUBLAS_ERROR(hipFree(dev_mat2));
    CHECK_CUBLAS_ERROR(hipFree(d_vector));
    CHECK_CUBLAS_ERROR(hipFree(true_vect));
    //fclose(stream);

}

void gen_init_matrix(float* dst, int rows, int cols, float min_val = 0, float max_val = 1) {
    if (NULL == dst)
        exit(-1);
    for (int r = 0; r < rows*cols; ++r) {
        dst[r] = (1.0f * (rand( )) / RAND_MAX) *(max_val - min_val) + min_val;
    }
}

void trans_mat(float* mat, int rows, int cols ) {
    float* tmp = (float*)(malloc)(rows*cols*sizeof(float));
    memcpy((void*)tmp, (void*)mat, rows*cols*sizeof(float));
    int count = 0;
    for (int i = 0; i < cols; ++i) {
        for (int j = 0; j < rows; ++j) {
            mat[count++] = tmp[j*cols + i];
        }
    }
    free(tmp);
}

void cpu_test(int rows, int cols, int dim, float* A, float* x, float* y, float alpha, float beta) {
    float* dst = (float*)malloc(rows*cols*(sizeof(float)));
    for (int i = 0; i < rows; ++i) {
//        float sum = 0.0f;
        for (int j = 0; j < cols; ++j){
            float sum = 0.0f;
            for (int k =0; k<dim;++k){
                sum += A[i*dim + k] * x[j*dim+k] * alpha;
            }
//        dst[i] = sum /*+ beta * y[i]*/;
//            printf("%2.6f\t", sum);
        }
    }
}

// void test( ) {
//     hipSetDevice(2);
//     int rows = 500000, cols = 500000, dim= 256;
//     float *mat1 = 0, *mat2 = 0, *mat3 = 0;
//     mat1 = (float*)malloc(rows*dim*(sizeof(float)));
//     mat2 = (float*)malloc(cols*dim*sizeof(float));
//     mat3 = (float*)malloc(alloc_size*alloc_size*sizeof(float));
//     gen_init_matrix(mat1, rows, dim, 0, 1);
//     gen_init_matrix(mat2, cols, dim , 0, 1);
//     gen_init_matrix(mat3, alloc_size, alloc_size, 0, 1);

// printf("cpu : %lf\n",(double)(t2-t1)/CLOCKS_PER_SEC);

//    trans_mat(mat1, rows, cols);
//    printf("mat1\n");
//    for (int i = 0; i < rows; ++i) {
//        for (int j = 0; j < dim; ++j)
//            printf("%f\t", mat1[i*dim + j]);
//        printf("\n");
//    }
//    printf("mat2\n");
//    for (int i = 0; i < cols; ++i) {
//        for (int j = 0; j < dim; ++j)
//          printf("%f\t", mat2[i*dim + j]);
//        printf("\n");
//    }
//    printf("\n");
//    printf("%f %f %f %d %d %d\n",*mat3,*mat1,*mat2,rows,cols,dim);    
    // clock_t t3 = clock();
    // printf("gpu : %lf\n",(double)(t3-t2)/CLOCKS_PER_SEC);

// }

// void readFeaturesCPU(char* fin, featM &ret)
// {
//     ret.mat = (float*)malloc(MAXSIZE*MAXDIM*(sizeof(float)));
//     std::ifstream featfile(fin);
//     if(!featfile.is_open()) {
//         std::cout << "can't open " << fin << std::endl;
//         exit(-1);
//     }
//     int linecnt = 0 , cnt = 0 , iDim = 0 ;
//     for (std::string strLine; std::getline(featfile, strLine); linecnt++)
//     {
//         std::istringstream iss(strLine);
//         if (HASFN)
//         {
//             std::string strFn;
//             iss >> strFn;
//         }
//         if (HASLABEL)
//         {
//             int tmp=0;
//             iss >> tmp;
//         }
//         iDim = 0;
//         while(iss >> ret.mat[cnt])
//             iDim+=1,cnt+=1;

   
//     }
//     ret.length = linecnt;
//     ret.dim = iDim;
//     printf("cnt == %d, linecnt*iDim = %d \n",cnt,linecnt * iDim);
//     assert(cnt == linecnt * iDim);


// }


int getDimCnt(char* buff)
{
  int i=0,dim=1 - HASFN - HASLABEL ;
  while(buff[i]!='\n')
  {
    if (buff[i]==' ' && buff[i-1]!=' ')
    {
        dim+=1;
    }
    i+=1;

  }
  return dim;

}
/*void readFeaturesGPU(char* fin, featM &ret)
{
    clock_t t1 = clock();
    
    std::ifstream inFile(fin, std::ios::binary | std::ios::in);
    inFile.seekg(0, std::ios::end);
    uint64_t nFileLen = inFile.tellg();
    std::vector<char> buf(nFileLen + 1ULL);
    std::cout << "Allocated " << nFileLen << " bytes" << std::endl;
    
    inFile.seekg(0, std::ios::beg);
    inFile.read(buf.data(), nFileLen);
    int dim = getDimCnt(buf.data());
    buf[nFileLen] = 0;
    std::cout << "Loaded " << nFileLen << " bytes" << std::endl;

    std::vector<uint64_t> lineBegins;
    lineBegins.push_back(0);
#pragma omp parallel
    {
#pragma omp for
    for (uint64_t i = 0; i < buf.size(); ++i)
    {
        if (buf[i] == '\n')
        {
#pragma omp critical
            lineBegins.push_back(i + 1);
            buf[i] = 0;
        }
    }
    }
    uint64_t nLineCnt = lineBegins.size() - 1;
    lineBegins.push_back(buf.size());
    std::cout << "Find " << nLineCnt << " lines" << std::endl;
    
    std::sort(lineBegins.begin(), lineBegins.end());
    std::cout << "Line begin positions sorted" << std::endl;


    
    printf("dimension of feature is %d\n",dim);

    ret.mat = (float*)malloc(nLineCnt*dim*(sizeof(float)));
    ret.length = nLineCnt;
    ret.dim = dim;
    float *features = ret.mat;
    // std::vector< std::array<float, 128> > features;
    // features.resize(nLineCnt);
    int stepsize=2000000*256/dim;
    for (uint64_t i = 0 ; i < nLineCnt;)
    {
        std::vector<uint64_t> begs;
        uint64_t iNext = std::min(nLineCnt, i + stepsize);
        char *pBase = buf.data() + lineBegins[i];
        // float *pOut = (float*)features[i].data();
        float *pOut = features + i*dim;
        std::cout << "Find index" << std::endl;
#pragma omp parallel for
        for (int j = i; j < iNext; ++j)
        {   
            // std::cout <<j << std::endl;
            char *pLine  = buf.data() + lineBegins[j];
            uint64_t jPos=0;
            if (HASFN)
            {               
                
                for (jPos = 0; pLine[jPos] != ' '; jPos++)
                    ;   
                if (HASLABEL)
                {
                    ++jPos;
                }
                for (; pLine[jPos] != ' '; jPos++)
                    ;
            }
                                                                



             // printf("%s",pLine[jPos + 1]);
            // for (int cc=jPos + 1; pLine[cc] != ' '; cc++)
            //  printf("%c",pLine[cc]);
            // printf("\n");
#pragma omp critical
            if (HASFN)
                begs.push_back(pLine + jPos + 1 - pBase);
            else
                begs.push_back(pLine  - pBase);

        }

        std::sort(begs.begin(), begs.end());
        std::cout << "Copy to GPU " << i << std::endl;

        
        StringsToFloatVectors(
                pBase,
                buf.data() + lineBegins[iNext] - pBase,
                begs,
                pOut,
                dim
                );
        i = iNext;

        
    }
    buf.clear();
    buf.shrink_to_fit();
    clock_t t2 = clock();
    printf("done reading! time:%f\n",(float)(t2-t1)/CLOCKS_PER_SEC);
    // for(int i=0;i<128;i++)
    //     printf("%f ",ret.mat[i]);
    // ret.mat = features;
    


}*/

void printMatrix( float* x, int rows , int dim)
{

    printf("the matrix is :\n");
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < dim; ++j)
           printf("%f ", x[i*dim + j]);
       printf("\n");
   }
}


int main(int argc, char** argv ) {

   
    return 0;
}
